#include "hip/hip_runtime.h"
#include<stdio.h>
#include"package_1_kernel.h"


__global__ void Child_1()
{
    printf("hello1\n");
}

__global__ void Parent_1()
{
    Child_1<<<1,1>>>();
    //hipDeviceSynchronize();
}

void Kernel_Function()
{
    Parent_1<<<1,1>>>();
    hipDeviceSynchronize();
}