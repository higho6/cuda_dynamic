#include "hip/hip_runtime.h"
#include<stdio.h>
#include<thrust/sort.h>
#include<thrust/execution_policy.h>
#include"package_2_kernel.h"


__global__ void Child_2()
{
    printf("hello2\n");
}

__global__ void Parent_2()
{

    hipDeviceSynchronize();
    printf("2parentkernel\n");
    hipDeviceSynchronize();
    Child_2<<<1,1>>>();

    hipDeviceSynchronize();

}

void Kernel_Function2()
{

    Parent_2<<<1,1>>>();
    hipDeviceSynchronize();
    printf("----out---\n");
    
    int array[5] = {5,3,4,8,9};
    int key[5] = {5,8,3,4,5};
    int *dev_array;
    int *dev_key;
    hipMalloc(&dev_array,5*sizeof(int));
    hipMalloc(&dev_key,5*sizeof(int));
    hipMemcpy(dev_array,array,5*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_key,key,5*sizeof(int),hipMemcpyHostToDevice);
    thrust::sort_by_key(thrust::device,dev_array,dev_array+5,dev_key);

    hipFree(dev_array);
    hipFree(dev_key);


}